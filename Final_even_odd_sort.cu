#include <stdio.h>
#include <hip/hip_runtime.h>



    __global__ void testKernel(int *in, int *out, int size)

    {

    bool oddeven=true;

    __shared__ bool swappedodd;

    __shared__ bool swappedeven;
    int temp,i,rem1;

    swappedodd=true;

    swappedeven=true;


    while(true)

    {

    if(oddeven==true)

    {

	     __syncthreads();

	     swappedodd=false;

	     __syncthreads();

	     if (threadIdx.y == 0) {

	      int idx=threadIdx.x;
	     if(idx<(size/2))

	  {

	   if (in[2*idx]>in[2*idx+1])

	   {

	   // swap(in[],in[2*idx+1]);
		temp= in[2*idx];
	    	      	   in[2*idx]=in[2*idx+1];
	    	      	   in[2*idx+1]=temp;
	    swappedodd=true;

	   }

	  }

	 }


	 __syncthreads();

    }

    else

    {

	     __syncthreads();

	     swappedeven=false;

	     __syncthreads();

	     if (threadIdx.y == 0) {

	      int idx=threadIdx.x;
	       if(idx<(size/2)-1)

	  {

	   if (in[2*idx+1]>in[2*idx+2])

	   {

	  //  swap(in[2*idx+1],in[2*idx+2]);
		temp= in[2*idx+1];
	    	      	   in[2*idx+1]=in[2*idx+2];
	    	      	   in[2*idx+2]=temp;
	    swappedeven=true;

	   }

	  }


	}

     __syncthreads();

    }

    if(!(swappedodd||swappedeven))

      break;

    oddeven=!oddeven;//switch mode of sorting

    }



    __syncthreads();



    int idx=threadIdx.x;

    if ( idx <size )

    out[idx]=in[idx];

    }


int main(void)
{
	int *a,*a_sorted,i;
	int *d_a,*d_sorted;
	int n=20;
	int size = sizeof(int)*n;

	hipMalloc((void**)&d_a,size);
	hipMalloc( (void**)&d_sorted, size);

   a=(int*)malloc(size);
   a_sorted=(int*)malloc(size);


	hipMalloc((void**)&d_sorted, size);

	printf("enter the unsorted numbers\n");
	for(i=0;i<n;i++)
	{
		scanf("%d",&a[i]);

	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);



	testKernel<<<1,n>>>(d_a, d_sorted,n);

	hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);


	hipMemcpy(a_sorted, d_sorted, size, hipMemcpyDeviceToHost);
	for (i=0;i<n;i++)
		{

		printf("%d",a_sorted[i]);
		printf("\t");

		}

free(a);

free(a_sorted);
hipFree(d_sorted);
hipFree(d_a);
}

