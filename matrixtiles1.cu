//Matrix multiplication using shared and non shared kernal




#include <hip/hip_runtime.h>
#include <stdio.h>

#include <math.h>

#define TILE_WIDTH 2

/*matrix multiplication kernels*/

 //non shared
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{

           // calculate thread id

           unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;

           unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

         for (int k = 0 ; k<WIDTH ; k++ )
         {
                  Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col] ;
          }
}



// shared
__global__ void MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{

        //Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele

          __shared__ float Mds [TILE_WIDTH][TILE_WIDTH] ;

           __shared__ float Nds [TILE_WIDTH][TILE_WIDTH] ;

         // calculate thread id
          unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
          unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
          float Pvalue = 0;
        for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ ) // m indicate number of phase
       {
            Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)]  ;
            Nds[threadIdx.y][threadIdx.x] =  Nd[ col+( m*TILE_WIDTH + threadIdx.y) * WIDTH ] ;
           __syncthreads() ; // for syncronizeing the threads


        for (int k = 0; k < TILE_WIDTH; ++k)
        {
                Pvalue += Mds[threadIdx.x][k] * Nds[k][threadIdx.y];
             
        }
       __syncthreads();
       }

         Pd[row*WIDTH + col] = Pvalue;
  }

// main routine
int main ()
{
   const int WIDTH = 6 ;
   float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],M_result_array_h[WIDTH][WIDTH] ;
   float *array1_d,*array2_d ,*M_result_array_d ; // device array  *result_array_d
   int i , j ;
  //input in host array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
     for (j = 0 ; j<WIDTH ; j++ )
     {
        array1_h[i][j] = 1 ;
        array2_h[i][j] = 2 ;
     }
  }

  //create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;

  hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;

  hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;



  //copy host array to device array; cudaMemcpy ( dest , source , WIDTH , direction )

  hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;

  hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;



  //allocating memory for resultent device array

 // cudaMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;

  hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) ) ;



  //calling kernal

  dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;

  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;

// Change if 0 to if 1 for running non shared code and make if 0 for shared memory code
#if 0

                MatrixMul<<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;

#endif

#if 1

               MatrixMulSh<<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;

#endif

  // all gpu function blocked till kernel is working
  //copy back result_array_d to result_array_h

  hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int),hipMemcpyDeviceToHost) ;

hipFree(array1_d);
hipFree(array2_d);
//cudaFree(result_array_d);
hipFree(M_result_array_d);

  //printf the result array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
      for ( j = 0 ; j < WIDTH ; j++ )
     {
        printf ("%f   ",M_result_array_h[i][j] ) ;
     }
 printf ("\n") ;
}
hipFree(M_result_array_h);
 //system("pause") ;
}
