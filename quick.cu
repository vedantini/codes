
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define MAX_THREADS 128

using namespace std;
const int threshold =25;

__global__ void bubble_sort(int *a, int left, int right)
{
	int temp;
	 for(int i=left;i<right;i++)
    	for(int j=i+1;j<=right;j++)
    		if(a[i]>a[j])
    		{
    			temp=a[i];
    			a[i]=a[j];
    			a[j]=temp;
    		}
}
__global__ void partition(int *a,int left,int right,int pivot,int *al,int *ah)
{
	int l,h;
	int size=(right-left+1);
	int k1=threadIdx.x*size+left;
	int k2=k1+size-1;

	if(threadIdx.x==MAX_THREADS-1)
		k2=right;
	l=h=k1;
	for(int i=k1;i<=k2;i++)
		{
			al[i]=ah[i]=-999;
		}
	for(int i=k1;i<=k2;i++)
	{
		if(a[i]<pivot)
		{
			al[l++]=a[i];
		}
		else
		{
			if(a[i]>pivot)
			{
				ah[h++]=a[i];
			}
		}
	}
}

void quicksort(int *a, const int left, const int right)
{
    
    if (right-left <= threshold)
    {
    	int *ad;
    	hipMalloc((void **)&ad,(right-left+1)*sizeof(int));
    	hipMemcpy(ad,a,(right-left+1)*sizeof(int),hipMemcpyHostToDevice);
        bubble_sort<<<1,1>>>(ad, left, right);
        hipMemcpy(a,ad,(right-left+1)*sizeof(int),hipMemcpyDeviceToHost);
        return;
    }
    int pivot = a[left];
    int size = (right-left+1)*sizeof(int);
    int *al,*ah,*ad;

    hipMalloc((void **)&ad,size);
    hipMalloc((void **)&al,size);
    hipMalloc((void **)&ah,size);
    hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    
    partition<<<1,MAX_THREADS>>>(ad,left,right,pivot,al,ah);
    int al_h[right-left+1],ah_h[right-left+1];
    hipMemcpy(al_h,al,size,hipMemcpyDeviceToHost);
    hipMemcpy(ah_h,ah,size,hipMemcpyDeviceToHost);
    int i=0,k=0;
    while(i<right-left+1)
    {
    	while(al_h[i]==-999 && i<right-left+1)
    			i++;
    	while(al_h[i]!=-999 && i<right-left+1)
    	{
    		al_h[k++]=al_h[i++];
    	}
    }
    
    quicksort(al_h,0,k-1);
    int p=left;
    int x=0;

        while(x<k)
        {
        	a[p++]=al_h[x++];

        }
        a[p]=pivot;
    	i=0;
    	k=0;
    while(i<right-left+1)
    {
      	while(ah_h[i]==-999 && i<right-left+1)
    		i++;
       	while(ah_h[i]!=-999 && i<right-left+1)
       	{
       		ah_h[k++]=ah_h[i++];
       	}
    }
    quicksort(ah_h,0,k-1);
    i=0;
    p++;
        while(i<k)
        {
        	a[p++]=ah_h[i++];
        }
}
int main()
{
	int len,x,flag,choice;
	cout<<"ENTER SIZE OF ARRAY :"<<endl;
	cin>>len;
	int input_data[len];
	cout<<"*********** MENU ***************"<<endl;
	cout<<"1. INPUT USER DATA "<<endl;
	cout<<"2. RANDOM GENERATOR "<<endl;
	cout<<"ENTER YOUR CHOICE.......... "<<endl;
	cin>>choice;
	switch(choice)
	{
		case 1:
			for (int i = 0 ; i < len ; i++)
    			{
				cout<<"ENTER ELEMENTS :" <<endl;
				cin>>input_data[i];
			}
    			cout<<"ORIGINAL ARRAY :"<<endl;
			for(int i=0;i<len;i++)
				cout<<input_data[i]<<"\t";
			cout<<endl;
    			quicksort(input_data,0,len-1);
    			cout<<"AFTER SORTING "<<endl;
			for(int i=0;i<len;i++)
	        		cout<<input_data[i]<<"\t";
			cout<<endl;
			break;
		case 2:
			for (int i = 0 ; i < len ; i++)
    			{

				    x=rand()%len;
    				    flag=0;
    				for(int j=0;j<i;j++)
    				{
    					if(input_data[j]==x)
    					{
    						i--;
    						flag=1;
    						break;
    					}
    				}
    				if(flag==0)
    				input_data[i]=x;
			}
    			cout<<"ORIGINAL ARRAY :"<<endl;
			for(int i=0;i<len;i++)
				cout<<input_data[i]<<"\t";
			cout<<endl;

    			quicksort(input_data,0,len-1);
    			cout<<"AFTER SORTING "<<endl;
			for(int i=0;i<len;i++)
	        		cout<<input_data[i]<<"\t";
			cout<<endl;
			break;
	}
return 0;
}

